﻿#include "Dynamics/RigidBody/PBDRigid/MultiHeightFieldRigidDetectionModule.h"
#include "Framework/Framework/ModuleTopology.h"
//#include "Dynamics/Sand/types.h"
//#include "Dynamics/RigidBody/RigidUtil.h"
//#include "Dynamics/RigidBody/RigidTimeIntegrationModule.h"
//#include "Dynamics/RigidBody/RKIntegrator.h"
#include <hip/device_functions.h>

#include <iostream>

#include "thrust/reduce.h"
#include "thrust/execution_policy.h"
#include "thrust/device_vector.h"

#include "Dynamics/RigidBody/ContactInfo.h"
#include "MultiHeightFieldRigidDetectionModule.h"

namespace PhysIKA
{
	MultiHeightFieldRigidDetectionModule::~MultiHeightFieldRigidDetectionModule()
	{
		m_contacts.release();
	}



	bool PhysIKA::MultiHeightFieldRigidDetectionModule::initializeImpl()
	{
		if (rigidContactDetector)
		{
			rigidContactDetector->varThreshold()->setValue(0.0001);
			rigidContactDetector->varContacts()->setValue(DeviceDArray<ContactInfo<double>>());
		}

		return true;
	}




	void MultiHeightFieldRigidDetectionModule::contactDetection(PBDSolver * solver, Real dt)
	{
		// Do contacts detection.
		int ncontacts = 0;
		for (int i = 0; i < heightfieldRigidDetectors.size(); ++i)
		{
			heightfieldRigidDetectors[i]->contactDetection(solver, dt);
			ncontacts += heightfieldRigidDetectors[i]->getContacts().size();
		}

		// Copy all contacts into one array.
		m_contacts.reserve(ncontacts);
		m_contacts.resize(0);
		int ntotal = 0;
		for (int i = 0; i < heightfieldRigidDetectors.size(); ++i)
		{
			auto& curContacts = heightfieldRigidDetectors[i]->getContacts();
			if (curContacts.size() > 0)
			{
				//cuSynchronize();

				// debug
				ntotal += curContacts.size();

				cuSafeCall(
					hipMemcpy(m_contacts.begin() + m_contacts.size(), curContacts.begin(), 
						sizeof(ContactInfo<double>)*curContacts.size(), hipMemcpyDeviceToDevice)
				);
				m_contacts.resize(m_contacts.size() + curContacts.size());
			}
		}

		// debug
		if(m_contacts.size()>0){
			HostDArray<ContactInfo<double>> hostContacts;
			hostContacts.resize(m_contacts.size());

			cuSafeCall(
				hipMemcpy(hostContacts.begin(), m_contacts.begin(), sizeof(ContactInfo<double>)* m_contacts.size(), hipMemcpyDeviceToHost)
			);

			hostContacts.release();
		}

		// Detect contact points between rigid bodies.
		if (rigidContactDetector)
			//if(false)
		{
			rigidContactDetector->doCollision();

			auto& contactArr = rigidContactDetector->varContacts()->getValue();
			if (contactArr.size() > 0)
			{
				int nContacts = m_contacts.size();
				m_contacts.resize(nContacts + contactArr.size());
				cuSafeCall(hipMemcpy(&(m_contacts[nContacts]), contactArr.begin(),
					sizeof(ContactInfo<double>)*contactArr.size(), hipMemcpyDeviceToDevice));

				//m_nContacts += contactArr.size();
			}
		}

		solver->setContactJoints(m_contacts, m_contacts.size());
	}


}